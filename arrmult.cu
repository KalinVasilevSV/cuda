
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <vector>
#include <iostream>

__global__
void add(float *a, float *b, float *c, long size){
    for(int i = 0; i< size;i++)
        c[i] = a[i] * b[i];
}

int main(void)
{
    long size = 3;
    float *a, *b,*c;

    hipMallocManaged(&a,sizeof(float)*size);
    hipMallocManaged(&b,sizeof(float)*size);
    hipMallocManaged(&c,sizeof(float)*size);

    for(int i=0;i<size;i++){
        a[i]=1.0;
        b[i]=2.0;
        c[i]=0.5;
    }

    add<<<1,1>>>(a,b,c,size);

    hipDeviceSynchronize();

    for(int i=0;i<size;i++){
        printf("%f ",c[i]);
    }

    hipFree(a);
    hipFree(b);
    hipFree(c);
}
